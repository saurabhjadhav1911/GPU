
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void square(float * d_out,float * d_in){
int idx=threadIdx.x;
float f=d_in[idx];
d_out[idx] = f*f;
}


int main(int argc,char ** argv){
const int N=64;
const int ARRAY_BYTES = N * sizeof(float);

float h_in[N];
for(int i=0;i<N;i++){
  h_in[i]=float(i);
  
}
float h_out[N];

float * d_in;
float * d_out;
hipMalloc((void **) &d_in, ARRAY_BYTES);
hipMalloc((void **) &d_out, ARRAY_BYTES);

hipMemcpy(d_in,h_in,ARRAY_BYTES,hipMemcpyHostToDevice);

square<<<1,N>>>(d_out,d_in);

hipMemcpy(h_out,d_out,ARRAY_BYTES,hipMemcpyDeviceToHost);

for(int i=0;i<N;i++){
  printf("%f   ",h_out[i]);
  printf("%f",h_in[i]);
  printf("\n");
}

hipFree(d_in);
hipFree(d_out);

return 0;
}