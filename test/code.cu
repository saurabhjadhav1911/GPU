
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 64

__global__ void square(float * d_out,float * d_in){
int idx=threadIdx.x;
float f=d_in[idx];
d_out[idx] = f*f/255;
}

void wrapper_square(float * d_out,float * d_in){

	square<<<1,N>>>(d_out,d_in);
}

int main(int argc,char ** argv){
const int ARRAY_BYTES = N * sizeof(float);

float h_in[N];
for(int i=0;i<N;i++){
  h_in[i]=float(i);
  
}
float h_out[N];

float * d_in;
float * d_out;
hipMalloc((void **) &d_in, ARRAY_BYTES);
hipMalloc((void **) &d_out, ARRAY_BYTES);

hipMemcpy(d_in,h_in,ARRAY_BYTES,hipMemcpyHostToDevice);

wrapper_square(d_out,d_in);

hipMemcpy(h_out,d_out,ARRAY_BYTES,hipMemcpyDeviceToHost);

for(int i=0;i<N;i++){
  printf("%f   ",h_out[i]);
  printf("%f",h_in[i]);
  printf("\n");
}

hipFree(d_in);
hipFree(d_out);

return 0;
}